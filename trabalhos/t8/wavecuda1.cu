/*
Adapted from fractal code for CS 4380 / CS 5351

Copyright (c) 2018, Texas State University. All rights reserved.

Redistribution and usage in source and binary form, with or without
modification, is only permitted for educational use.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
Revision history:
20190610   andreainfufsm   Replaced the function to calculate the color of each pixel
20190615   rafael-v        Added CUDA to compute each frame on a different thread
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <sys/time.h>
#include <math.h>

__global__ void computeFrame(unsigned char* pic, int frames, int width) {

  int idx = threadIdx.x;
  int stride = blockDim.x;
  for (int frame = idx; frame < frames; frame += stride) {
    for (int row = 0; row < width; row++) {
      for (int col = 0; col < width; col++) {
        float fx = col - 1024/2;
        float fy = row - 1024/2;
        float d = sqrtf( fx * fx + fy * fy );
        unsigned char color = (unsigned char) (160.0f + 127.0f *
                                              cos(d/10.0f - frame/7.0f) /
                                              (d/50.0f + 1.0f));

        pic[frame * width * width + row * width + col] = (unsigned char) color;
      }
    }
  }
}

int main(int argc, char *argv[]) {

  // check cuda device
  int devCount;
  hipGetDeviceCount(&devCount);
  if (devCount < 1) {fprintf(stderr, "error: there are no cuda devices\n"); exit(-1);}

  // check command line
  if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
  int width = atoi(argv[1]);
  if (width < 100) {fprintf(stderr, "error: frame_width must be at least 100\n"); exit(-1);}
  int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("computing %d frames of %d by %d picture\n", frames, width, width);

  // allocate picture array
  unsigned char* pic;
  hipMallocManaged(&pic, frames * width * width * sizeof(unsigned char));

  // get the maximum number of threads per block supported by the cuda device
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  int maxThreadsPerBlock = devProp.maxThreadsPerBlock;

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);
  
  // run kernel on the GPU using the number of threads equal to the number of frames, or the maximum supported
  int numThreads = (frames > maxThreadsPerBlock) ? maxThreadsPerBlock : frames;
  computeFrame<<<1, numThreads>>>(pic, frames, width);

  // wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  // free memory
  hipFree(pic);

  return 0;
}
